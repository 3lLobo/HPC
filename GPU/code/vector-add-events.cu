#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include "timer.h"
#include <iostream>

using namespace std;

/* Utility function, use to do error checking.

   Use this function like this:

   checkCudaCall(hipMalloc((void **) &deviceRGB, imgS * sizeof(color_t)));

   And to check the result of a kernel invocation:

   checkCudaCall(hipGetLastError());
*/
static void checkCudaCall(hipError_t result) {
    if (result != hipSuccess) {
        cerr << "cuda error: " << hipGetErrorString(result) << endl;
        exit(1);
    }
}


__global__ void vectorAddKernel(int opr, float* A, float* B, float* Result) {
    int i = (blockIdx.x * blockDim.x) + threadIdx.x;

    if (opr == 0) {
    	Result[i] = A[i] + B[i];
    }
    else if (opr == 1){
	Result[i] = A[i] - B[i];
    }
    else if (opr == 2){
	Result[i] = A[i] * B[i];
    }
    else
    {
	if (A[i] > 0) {Result[i] = (float) A[i] / B[i];}
        else { Result[i] = 0; } 
    }

}

void vectorAddCuda(int opr, int n, float* a, float* b, float* result) {
    int threadBlockSize = 512;

    // allocate the vectors on the GPU
    float* deviceA = NULL;
    checkCudaCall(hipMalloc((void **) &deviceA, n * sizeof(float)));
    if (deviceA == NULL) {
        cout << "could not allocate memory!" << endl;
        return;
    }
    float* deviceB = NULL;
    checkCudaCall(hipMalloc((void **) &deviceB, n * sizeof(float)));
    if (deviceB == NULL) {
        checkCudaCall(hipFree(deviceA));
        cout << "could not allocate memory!" << endl;
        return;
    }
    float* deviceResult = NULL;
    checkCudaCall(hipMalloc((void **) &deviceResult, n * sizeof(float)));
    if (deviceResult == NULL) {
        checkCudaCall(hipFree(deviceA));
        checkCudaCall(hipFree(deviceB));
        cout << "could not allocate memory!" << endl;
        return;
    }

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // copy the original vectors to the GPU
    checkCudaCall(hipMemcpy(deviceA, a, n*sizeof(float), hipMemcpyHostToDevice));
    checkCudaCall(hipMemcpy(deviceB, b, n*sizeof(float), hipMemcpyHostToDevice));

    // execute kernel
    hipEventRecord(start, 0);
    vectorAddKernel<<<ceil((double)n/threadBlockSize), threadBlockSize>>>(opr, deviceA, deviceB, deviceResult);
    hipEventRecord(stop, 0);

    // check whether the kernel invocation was successful
    checkCudaCall(hipGetLastError());

    // copy result back
    checkCudaCall(hipMemcpy(result, deviceResult, n * sizeof(float), hipMemcpyDeviceToHost));

    checkCudaCall(hipFree(deviceA));
    checkCudaCall(hipFree(deviceB));
    checkCudaCall(hipFree(deviceResult));

    // print the time the kernel invocation took, without the copies!
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    
    cout << "kernel invocation took \t\t" << elapsedTime << " milliseconds" << endl;
}


int main(int argc, char* argv[]) {
    int arr[3] = { 65536, 655360, 1000000 };
    const char *ops[] = {"Adding", "Subtracting", "Multiply", "Divide"};
    int opr;
    
    for (int j = 0; j<4; j++){
	opr = j;   
	for (int i = 0; i < 3; i++){
	    int n = arr[i];
            float* a = new float[n];
            float* b = new float[n];
            float* result = new float[n];
            float* result_s = new float[n];

            if (argc > 1) n = atoi(argv[1]);
            cout << ops[j] << " two vectors of " << n << " integer elements." << endl;
            // initialize the vectors.
            for(int i=0; i<n; i++) {
                a[i] = i;
                b[i] = i;
            }

            vectorAddCuda(opr, n, a, b, result);

            cout << "Done!" << endl;

            delete[] a;
            delete[] b;
            delete[] result;
            delete[] result_s;

        }
    }

    return 0;
}
